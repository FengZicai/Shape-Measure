#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_helper.h"
//n<=4096, m<=1024
__global__ void approxmatch(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,float * __restrict__ match,float * temp){
	float * remainL=temp+blockIdx.x*(n+m)*2, * remainR=temp+blockIdx.x*(n+m)*2+n,*ratioL=temp+blockIdx.x*(n+m)*2+n+m,*ratioR=temp+blockIdx.x*(n+m)*2+n+m+n;
	float multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	const int Block=1024;
	__shared__ float buf[Block*4];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			float level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				float x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				float suml=1e-9f;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						float x2=xyz2[i*m*3+l0*3+l*3+0];
						float y2=xyz2[i*m*3+l0*3+l*3+1];
						float z2=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+0]=x2;
						buf[l*4+1]=y2;
						buf[l*4+2]=z2;
						buf[l*4+3]=remainR[l0+l];
					}
					__syncthreads();
					for (int l=0;l<lend;l++){
						float x2=buf[l*4+0];
						float y2=buf[l*4+1];
						float z2=buf[l*4+2];
						float d=level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1));
						float w=__expf(d)*buf[l*4+3];
						suml+=w;
					}
					__syncthreads();
				}
				if (k<n)
					ratioL[k]=remainL[k]/suml;
			}
			/*for (int k=threadIdx.x;k<n;k+=gridDim.x){
				float x1=xyz1[i*n*3+k*3+0];
				float y1=xyz1[i*n*3+k*3+1];
				float z1=xyz1[i*n*3+k*3+2];
				float suml=1e-9f;
				for (int l=0;l<m;l++){
					float x2=xyz2[i*m*3+l*3+0];
					float y2=xyz2[i*m*3+l*3+1];
					float z2=xyz2[i*m*3+l*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*remainR[l];
					suml+=w;
				}
				ratioL[k]=remainL[k]/suml;
			}*/
			__syncthreads();
			for (int l0=0;l0<m;l0+=blockDim.x){
				int l=l0+threadIdx.x;
				float x2=0,y2=0,z2=0;
				if (l<m){
					x2=xyz2[i*m*3+l*3+0];
					y2=xyz2[i*m*3+l*3+1];
					z2=xyz2[i*m*3+l*3+2];
				}
				float sumr=0;
				for (int k0=0;k0<n;k0+=Block){
					int kend=min(n,k0+Block)-k0;
					for (int k=threadIdx.x;k<kend;k+=blockDim.x){
						buf[k*4+0]=xyz1[i*n*3+k0*3+k*3+0];
						buf[k*4+1]=xyz1[i*n*3+k0*3+k*3+1];
						buf[k*4+2]=xyz1[i*n*3+k0*3+k*3+2];
						buf[k*4+3]=ratioL[k0+k];
					}
					__syncthreads();
					for (int k=0;k<kend;k++){
						float x1=buf[k*4+0];
						float y1=buf[k*4+1];
						float z1=buf[k*4+2];
						float w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*buf[k*4+3];
						sumr+=w;
					}
					__syncthreads();
				}
				if (l<m){
					sumr*=remainR[l];
					float consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
					ratioR[l]=consumption*remainR[l];
					remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
				}
			}
			/*for (int l=threadIdx.x;l<m;l+=blockDim.x){
				float x2=xyz2[i*m*3+l*3+0];
				float y2=xyz2[i*m*3+l*3+1];
				float z2=xyz2[i*m*3+l*3+2];
				float sumr=0;
				for (int k=0;k<n;k++){
					float x1=xyz1[i*n*3+k*3+0];
					float y1=xyz1[i*n*3+k*3+1];
					float z1=xyz1[i*n*3+k*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*ratioL[k];
					sumr+=w;
				}
				sumr*=remainR[l];
				float consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
				ratioR[l]=consumption*remainR[l];
				remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
			}*/
			__syncthreads();
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				float x1=0,y1=0,z1=0;
				if (k<n){
					x1=xyz1[i*n*3+k*3+0];
					y1=xyz1[i*n*3+k*3+1];
					z1=xyz1[i*n*3+k*3+2];
				}
				float suml=0;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						buf[l*4+0]=xyz2[i*m*3+l0*3+l*3+0];
						buf[l*4+1]=xyz2[i*m*3+l0*3+l*3+1];
						buf[l*4+2]=xyz2[i*m*3+l0*3+l*3+2];
						buf[l*4+3]=ratioR[l0+l];
					}
					__syncthreads();
					float rl=ratioL[k];
					if (k<n){
						for (int l=0;l<lend;l++){
							float x2=buf[l*4+0];
							float y2=buf[l*4+1];
							float z2=buf[l*4+2];
							float w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*rl*buf[l*4+3];
							match[i*n*m+(l0+l)*n+k]+=w;
							suml+=w;
						}
					}
					__syncthreads();
				}
				if (k<n)
					remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			/*for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float x1=xyz1[i*n*3+k*3+0];
				float y1=xyz1[i*n*3+k*3+1];
				float z1=xyz1[i*n*3+k*3+2];
				float suml=0;
				for (int l=0;l<m;l++){
					float x2=xyz2[i*m*3+l*3+0];
					float y2=xyz2[i*m*3+l*3+1];
					float z2=xyz2[i*m*3+l*3+2];
					float w=expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)))*ratioL[k]*ratioR[l];
					match[i*n*m+l*n+k]+=w;
					suml+=w;
				}
				remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}*/
			__syncthreads();
		}
	}
}

__global__ void approxmatch_legacy(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,float * __restrict__ match){
	const int MaxN=4096,MaxM=1024;
	__shared__ float remainL[MaxN],remainR[MaxM],ratioR[MaxM],ratioL[MaxN];
	__shared__ int listR[MaxM],lc;
	float multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			float level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			if (threadIdx.x==0){
				lc=0;
				for (int k=0;k<m;k++)
					if (remainR[k]>0)
						listR[lc++]=k;
			}
			__syncthreads();
			int _lc=lc;
			for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float suml=1e-9f;
				float x1=xyz1[(i*n+k)*3+0];
				float y1=xyz1[(i*n+k)*3+1];
				float z1=xyz1[(i*n+k)*3+2];
				//for (int l=0;l<m;l++){
				for (int _l=0;_l<_lc;_l++){
					int l=listR[_l];
					float x2=xyz2[(i*m+l)*3+0]-x1;
					float y2=xyz2[(i*m+l)*3+1]-y1;
					float z2=xyz2[(i*m+l)*3+2]-z1;
					float w=expf(level*(x2*x2+y2*y2+z2*z2))*remainR[l];
					suml+=w;
				}
				ratioL[k]=remainL[k]/suml;
			}
			__syncthreads();
			//for (int k=threadIdx.x;k<m;k+=blockDim.x){
			for (int _k=threadIdx.x;_k<lc;_k+=blockDim.x){
				int k=listR[_k];
				float sumr=0;
				float x2=xyz2[(i*m+k)*3+0];
				float y2=xyz2[(i*m+k)*3+1];
				float z2=xyz2[(i*m+k)*3+2];
				for (int l=0;l<n;l++){
					float x1=xyz1[(i*n+l)*3+0]-x2;
					float y1=xyz1[(i*n+l)*3+1]-y2;
					float z1=xyz1[(i*n+l)*3+2]-z2;
					float w=expf(level*(x1*x1+y1*y1+z1*z1))*ratioL[l];
					sumr+=w;
				}
				sumr*=remainR[k];
				float consumption=fminf(remainR[k]/(sumr+1e-9f),1.0f);
				ratioR[k]=consumption*remainR[k];
				remainR[k]=fmaxf(0.0f,remainR[k]-sumr);
			}
			__syncthreads();
			for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float suml=0;
				float x1=xyz1[(i*n+k)*3+0];
				float y1=xyz1[(i*n+k)*3+1];
				float z1=xyz1[(i*n+k)*3+2];
				for (int _l=0;_l<_lc;_l++){
					int l=listR[_l];
					float x2=xyz2[(i*m+l)*3+0]-x1;
					float y2=xyz2[(i*m+l)*3+1]-y1;
					float z2=xyz2[(i*m+l)*3+2]-z1;
					float w=expf(level*(x2*x2+y2*y2+z2*z2))*ratioL[k]*ratioR[l];
					match[i*n*m+l*n+k]+=w;
					suml+=w;
				}
				remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			__syncthreads();
		}
	}
}

void approxmatchLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,float * match, float * temp){
	approxmatch<<<32,512>>>(b,n,m,xyz1,xyz2,match,temp);
	
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

__global__ void matchcost(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * __restrict__ out){
	__shared__ float allsum[512];
	const int Block=256;
	__shared__ float buf[Block*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		float subsum=0;
		for (int k0=0;k0<m;k0+=Block){
			int endk=min(m,k0+Block);
			for (int k=threadIdx.x;k<(endk-k0)*3;k+=blockDim.x){
				buf[k]=xyz2[i*m*3+k0*3+k];
			}
			__syncthreads();
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				float x1=xyz1[(i*n+j)*3+0];
				float y1=xyz1[(i*n+j)*3+1];
				float z1=xyz1[(i*n+j)*3+2];
				for (int k=0;k<endk-k0;k++){
					//float x2=xyz2[(i*m+k)*3+0]-x1;
					//float y2=xyz2[(i*m+k)*3+1]-y1;
					//float z2=xyz2[(i*m+k)*3+2]-z1;
					float x2=buf[k*3+0]-x1;
					float y2=buf[k*3+1]-y1;
					float z2=buf[k*3+2]-z1;
					float d=sqrtf(x2*x2+y2*y2+z2*z2);
					subsum+=match[i*n*m+(k0+k)*n+j]*d;
				}
			}
			__syncthreads();
		}
		allsum[threadIdx.x]=subsum;
		for (int j=1;j<blockDim.x;j<<=1){
			__syncthreads();
			if ((threadIdx.x&j)==0 && threadIdx.x+j<blockDim.x){
				allsum[threadIdx.x]+=allsum[threadIdx.x+j];
			}
		}
		if (threadIdx.x==0)
			out[i]=allsum[0];
		__syncthreads();
	}
}

void matchcostLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,const float * match,float * out){
	matchcost<<<32,512>>>(b,n,m,xyz1,xyz2,match,out);

	CUDA_CHECK(hipGetLastError());
}

__global__ void matchcostgrad2(int b,
	int n,
	int m,
	const float * __restrict__ xyz1,
	const float * __restrict__ xyz2,
	const float * __restrict__ match,
	float * grad2)
{
	__shared__ float sum_grad[256*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		int kbeg=m*blockIdx.y/gridDim.y;
		int kend=m*(blockIdx.y+1)/gridDim.y;
		for (int k=kbeg;k<kend;k++){
			float x2=xyz2[(i*m+k)*3+0];
			float y2=xyz2[(i*m+k)*3+1];
			float z2=xyz2[(i*m+k)*3+2];
			float subsumx=0,subsumy=0,subsumz=0;
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				float x1=x2-xyz1[(i*n+j)*3+0];
				float y1=y2-xyz1[(i*n+j)*3+1];
				float z1=z2-xyz1[(i*n+j)*3+2];
				float d=match[i*n*m+k*n+j]/fmaxf(sqrtf(x1*x1+y1*y1+z1*z1),1e-20f);
				subsumx+=x1*d;
				subsumy+=y1*d;
				subsumz+=z1*d;
			}
			sum_grad[threadIdx.x*3+0]=subsumx;
			sum_grad[threadIdx.x*3+1]=subsumy;
			sum_grad[threadIdx.x*3+2]=subsumz;
			for (int j=1;j<blockDim.x;j<<=1){
				__syncthreads();
				int j1=threadIdx.x;
				int j2=threadIdx.x+j;
				if ((j1&j)==0 && j2<blockDim.x){
					sum_grad[j1*3+0]+=sum_grad[j2*3+0];
					sum_grad[j1*3+1]+=sum_grad[j2*3+1];
					sum_grad[j1*3+2]+=sum_grad[j2*3+2];
				}
			}
			if (threadIdx.x==0){
				grad2[(i*m+k)*3+0]=sum_grad[0];
				grad2[(i*m+k)*3+1]=sum_grad[1];
				grad2[(i*m+k)*3+2]=sum_grad[2];
			}
			__syncthreads();
		}
	}
}

__global__ void matchcostgrad1(int b,
	int n,
	int m,
	const float * __restrict__ xyz1,
	const float * __restrict__ xyz2,
	const float * __restrict__ match,
	float * grad1)
{
	for (int i = blockIdx.x; i < b; i += gridDim.x){
		for (int l = threadIdx.x; l < n; l += blockDim.x){
			float x1 = xyz1[i*n*3+l*3+0];
			float y1 = xyz1[i*n*3+l*3+1];
			float z1 = xyz1[i*n*3+l*3+2];
			float dx=0,dy=0,dz=0;
			for (int k = 0; k < m; k++){
				float x2 = x1 - xyz2[i*m*3+k*3+0];
				float y2 = y1 - xyz2[i*m*3+k*3+1];
				float z2 = z1 - xyz2[i*m*3+k*3+2];
				float d  = match[i*n*m+k*n+l] * fmaxf(sqrtf(x2*x2+y2*y2+z2*z2),1e-20f);

				dx += x2 * d; 
				dy += y2 * d;
				dz += z2 * d;
			}
			grad1[i*n*3+l*3+0] = dx;
			grad1[i*n*3+l*3+1] = dy;
			grad1[i*n*3+l*3+2] = dz;
		}
	}
}

void matchcostgradLauncher(int b,
	int n,
	int m,
	const float * xyz1,
	const float * xyz2,
	const float * match,
	float * grad1,
	float * grad2)
	{
	matchcostgrad1<<<32,512>>>(b,n,m,xyz1,xyz2,match,grad1);
	matchcostgrad2<<<dim3(32,32),256>>>(b,n,m,xyz1,xyz2,match,grad2);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("error in chamfer distance updateOutput: %s\n", hipGetErrorString(err));
}